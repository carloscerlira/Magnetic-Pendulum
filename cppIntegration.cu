#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "template.cuh"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

extern int res;

__device__ float map(const float& x, float x11, float x12, float x21, float x22) {
	float a = x21 + ((x22 - x21) / (x12 - x11)) * (x - x11);
	return a;
}

__device__ float2 test(float x, float y, float vx, float vy, float3& Px, float3& Py) {
	float2 a; float A, B, C, b, c, h, m, M, g, L; float sum_x; float sum_y;
	
	b = 0.1; m = 30.0; 
	g = 981.0 ; L = 17.0;

	c = 3.3746; h = sqrt(.25);
	M = 1.0;

	A = 3.3746;
	B = (3/2)*(b/m);
	C = (3/2)*(g/L);

	float& Px1 = Px.x; float& Px2 = Px.y; float& Px3 = Px.z;
	float& Py1 = Py.x; float& Py2 = Py.y; float& Py3 = Py.z;

	float s1_x, s2_x, s3_x;
	float s1_y, s2_y, s3_y;

	float r1_x, r2_x, r3_x;
	float r1_y, r2_y, r3_y;

	s1_x = Px1 - x;
	s2_x = Px2 - x;
	s3_x = Px3 - x;

	s1_y = Py1 - y;
	s2_y = Py2 - y;
	s3_y = Py3 - y;

	r1_x = s1_x / powf(powf(s1_x, 2.0) + powf(s1_y, 2.0) + powf(h, 2.0), 5.0 / 2.0);
	r2_x = s2_x / powf(powf(s2_x, 2.0) + powf(s2_y, 2.0) + powf(h, 2.0), 5.0 / 2.0);
	r3_x = s3_x / powf(powf(s3_x, 2.0) + powf(s3_y, 2.0) + powf(h, 2.0), 5.0 / 2.0);

	r1_y = s1_y / powf(powf(s1_x, 2.0) + powf(s1_y, 2.0) + powf(h, 2.0), 5.0 / 2.0);
	r2_y = s2_y / powf(powf(s2_x, 2.0) + powf(s2_y, 2.0) + powf(h, 2.0), 5.0 / 2.0);
	r3_y = s3_y / powf(powf(s3_x, 2.0) + powf(s3_y, 2.0) + powf(h, 2.0), 5.0 / 2.0);

	sum_x = r1_x + r2_x + r3_x;
	sum_y = r1_y + r2_y + r3_y;

	a.x = A * sum_x - B * vx - C * x;
	a.y = A * sum_y - B * vy - C * y;

	return a;
}

__device__ float path(float& x0, float& y0, float t, int n, float3& Px, float3& Py) {
	float2 p1; float2 r1;
	float2 p2; float2 r2;

	p1.x = x0; p1.y = y0;
	r1.x = 0; r1.y = 0;

	float dt = t / n;

	for (int i = 0; i < n; i++) {
		float dt2 = powf(dt, 2.0);

		if (i == 0) {
			float2& x2 = p1; float2& v2 = r1;
			float2 x3; float2 v3;
			float2 a2 = test(x2.x, x2.y, v2.x, v2.y, Px, Py);

			x3.x = x2.x + v2.x * dt + (1.0 / 2.0)* a2.x *dt2;
			x3.y = x2.y + v2.y * dt + (1.0 / 2.0)* a2.y *dt2;

			v3.x = v2.x + a2.x * dt;
			v3.y = v2.y + a2.y * dt;

			p2 = x3;
			r2 = v3;

			continue;
		}
		float2& x1 = p1; float2& x2 = p2; float2 x3;
		float2& v1 = r1; float2& v2 = r2; float2 v3; float2 v4;
		float2 a1; float2 a2; float2 a3;
		
		a1 = test(x1.x, x1.y, v1.x, v1.y, Px, Py);
		a2 = test(x2.x, x2.y, v2.x, v2.y, Px, Py);

		x3.x = x2.x + v2.x*dt + (2.0 / 3.0)* a2.x *dt2 - (1.0 / 6.0)* a1.x *dt2;
		x3.y = x2.y + v2.y*dt + (2.0 / 3.0)* a2.y *dt2 - (1.0 / 6.0)* a1.y *dt2;

		v3.x = v2.x + (3.0 / 2.0)* a2.x *dt - (1.0 / 2.0)* a1.x *dt;
		v3.y = v2.y + (3.0 / 2.0)* a2.y *dt - (1.0 / 2.0)* a1.y *dt;

		a3 = test(x3.x, x3.y, v3.x, v3.y, Px, Py);
		
		v4.x = v2.x + (5.0 / 12.0)* a3.x *dt + (2.0 / 3.0)* a2.x *dt - (1.0 / 12.0)* a1.x *dt;
		v4.y = v2.y + (5.0 / 12.0)* a3.y *dt + (2.0 / 3.0)* a2.y *dt - (1.0 / 12.0)* a1.y *dt;

		p1 = p2;
		r1 = r2;

		p2 = x3;
		r2 = v4;
	}

	float rx_1, rx_2, rx_3;
	rx_1 = p2.x - Px.x;
	rx_2 = p2.x - Px.y;
	rx_3 = p2.x - Px.z;

	float ry_1, ry_2, ry_3;
	ry_1 = p2.y - Py.x;
	ry_2 = p2.y - Py.y;
	ry_3 = p2.y - Py.z;

	float dis[3];
	dis[0] = powf(powf(rx_1, 2.0) + powf(ry_1, 2.0), 1.0 / 2.0);
	dis[1] = powf(powf(rx_2, 2.0) + powf(ry_2, 2.0), 1.0 / 2.0);
	dis[2] = powf(powf(rx_3, 2.0) + powf(ry_3, 2.0), 1.0 / 2.0);

	float dmin = 10000;
	float index;

	for (int i = 0; i < 3; i++)
	{
		if (dis[i] < dmin) {
			dmin = dis[i];
			index = i*1.0;
		}
	}
	return index;
}

__global__ void vectorAdd(float *A, int numElements)
{
	__shared__ float3 Px, Py;
	float pi = 3.141592;
	Px.x = 1*cosf(pi/3); Px.y = 1*cosf(pi); Px.z = 1*cosf(-pi/3);
	Py.x = 1*sinf(pi/3); Py.y = 1*sinf(pi); Py.z = 1*sinf(-pi/3);

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		float s1 = sqrtf(numElements);
		int off = i / s1; float x = i - off * s1; float y = off;
		float a, b;
		a = map(x, 0.0, s1, -2.5, 2.5);
		b = map(y, 0.0, s1, -2.5, 2.5);
		float v;
		v = path(a, b, 10.0, 1000, Px, Py);
		A[i*3] = a; A[i*3 + 1] = b; A[i*3 + 2] = v;
	}
}

extern "C" float* runTest()
{
	hipError_t err = hipSuccess;
	int numElements = pow(res, 2.0);
	size_t size = 3 * numElements * sizeof(float);

	float *h_A = (float *)malloc(size);
	float *d_A = NULL;
	err = hipMalloc((void **)&d_A, size);

	int threadsPerBlock = 500;
	int blocksPerGrid = (numElements + threadsPerBlock) / threadsPerBlock;

	vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, numElements);
	err = hipGetLastError();

	err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
	err = hipFree(d_A);
	return h_A;
}
